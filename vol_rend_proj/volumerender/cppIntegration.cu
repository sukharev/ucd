#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2007 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and
 * international Copyright laws.  Users and possessors of this source code
 * are hereby granted a nonexclusive, royalty-free license to use this code
 * in individual and commercial software.
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
 * OF USE, DATA OR PROFITS,  WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
 * OR OTHER TORTIOUS ACTION,  ARISING OUT OF OR IN CONNECTION WITH THE USE
 * OR PERFORMANCE OF THIS SOURCE CODE.
 *
 * U.S. Government End Users.   This source code is a "commercial item" as
 * that term is defined at  48 C.F.R. 2.101 (OCT 1995), consisting  of
 * "commercial computer  software"  and "commercial computer software
 * documentation" as such terms are  used in 48 C.F.R. 12.212 (SEPT 1995)
 * and is provided to the U.S. Government only as a commercial end item.
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the
 * source code with only those rights set forth herein.
 *
 * Any use of this source code in individual and commercial software must
 * include, in the user documentation and internal comments to the code,
 * the above Disclaimer and U.S. Government End Users Notice.
 */

/* Example of integrating CUDA functions into an existing 
 * application / framework.
 * Host part of the device code.
 * Compiled with Cuda compiler.
 */

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <iostream>
// includes, project
#include <cutil_inline.h>

// includes, kernels
#include <cppIntegration_kernel.cu>
//
////////////////////////////////////////////////////////////////////////////////
// declaration, forward

//extern "C" void
//computeGold(char* reference, char* idata, const unsigned int len);
//extern "C" void
//computeGold2(int2* reference, int2* idata, const unsigned int len);

extern "C"
void corr_cpu( float* C, /* out - correlations */
			  float* A, /* array of vectors stored in 1D array */
			  float* B, /* reference vector */
			  int length, /* length of vector */ 
			  int num /* number of vectors */);


// Host code
// N number of timesteps
// M volume size
extern "C" void
runTest1(int N, int M, float* h_A, float* h_B, float* h_C, int max_num_threads )
{
	// Allocate vectors in device memory
	float* h_Cref = new float[M];
	
	size_t sizeA = N * M * sizeof(float);
	size_t sizeB = N * 1 * sizeof(float);
	size_t sizeC = M * sizeof(float);
	
	float* d_A;
	CUDA_SAFE_CALL(hipMalloc((void**)&d_A, sizeA));
	float* d_B;
	CUDA_SAFE_CALL(hipMalloc((void**)&d_B, sizeB));
	float* d_C;
	CUDA_SAFE_CALL(hipMalloc((void**)&d_C, sizeC));
	// Copy vectors from host memory to device memory
	// h_A and h_B are input vectors stored in host memory
	CUDA_SAFE_CALL(hipMemcpy(d_A, h_A, sizeA, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_B, h_B, sizeB, hipMemcpyHostToDevice));
	// Invoke kernel
	//int threadsPerBlock = 198;//256;
	//int blocksPerGrid = 3240;//(N + threadsPerBlock-1)/ threadsPerBlock;
	int threadsPerBlock = max_num_threads;
	int blocksPerGrid = (int)M/(int)threadsPerBlock + 2;

	//VecAdd<<<blocksPerGrid, threadsPerBlock>>>(N,d_A, d_B, d_C);
	corr_cuda<<<blocksPerGrid, threadsPerBlock>>>( d_C, d_A, d_B, N, M);
	
	// check if kernel execution generated and error
    CUT_CHECK_ERROR("Kernel execution failed");
    
    hipDeviceSynchronize();
    
    // Copy result from device memory to host memory
	// h_C contains the result in host memory
	CUDA_SAFE_CALL(hipMemcpy(h_C, d_C, sizeC, hipMemcpyDeviceToHost));
	
	bool success = true;
	/*
    // compute reference solutions
    corr_cpu( h_Cref, h_A, h_B, N, M);
	
	// check result
    for( int i = 0; i < M; i++ )
    {
        if( abs(h_Cref[i] - h_C[i]) > SMALL_ERR)
            success = false;
    }
    
    if(!success){
		for( int i = 0; i < M; i++ )
		{
			std::cout << h_Cref[i] << ", ";
		}
		std::cout << std::endl;
    }
   
    printf("Test %s\n", success ? "PASSED" : "FAILED");
    */ 
	// Free device memory
	CUDA_SAFE_CALL(hipFree(d_A));
	CUDA_SAFE_CALL(hipFree(d_B));
	CUDA_SAFE_CALL(hipFree(d_C));
	
	delete [] h_Cref;
}