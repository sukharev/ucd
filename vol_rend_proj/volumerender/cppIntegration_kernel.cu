#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2007 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and
 * international Copyright laws.  Users and possessors of this source code
 * are hereby granted a nonexclusive, royalty-free license to use this code
 * in individual and commercial software.
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
 * OF USE, DATA OR PROFITS,  WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
 * OR OTHER TORTIOUS ACTION,  ARISING OUT OF OR IN CONNECTION WITH THE USE
 * OR PERFORMANCE OF THIS SOURCE CODE.
 *
 * U.S. Government End Users.   This source code is a "commercial item" as
 * that term is defined at  48 C.F.R. 2.101 (OCT 1995), consisting  of
 * "commercial computer  software"  and "commercial computer software
 * documentation" as such terms are  used in 48 C.F.R. 12.212 (SEPT 1995)
 * and is provided to the U.S. Government only as a commercial end item.
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the
 * source code with only those rights set forth herein.
 *
 * Any use of this source code in individual and commercial software must
 * include, in the user documentation and internal comments to the code,
 * the above Disclaimer and U.S. Government End Users Notice.
 */

/* Example of integrating CUDA functions into an existing 
 * application / framework.
 * Device code.
 */

#ifndef _CPP_INTEGRATION_KERNEL_H_
#define _CPP_INTEGRATION_KERNEL_H_
#include "matrixMul.h"

#define CHECK_BANK_CONFLICTS 0
#if CHECK_BANK_CONFLICTS
#define AS(i, j) CUT_BANK_CHECKER(((float*)&As[0][0]), (BLOCK_SIZE * i + j))
#define BS(i, j) CUT_BANK_CHECKER(((float*)&Bs[0][0]), (BLOCK_SIZE * i + j))
#else
#define AS(i, j) As[i][j]
#define BS(i, j) Bs[i][j]
#endif

///////////////////////////////////////////////////////////////////////////////
//! Simple test kernel for device functionality
//! @param g_odata  memory to process (in and out)
///////////////////////////////////////////////////////////////////////////////
__global__ void
kernel( int* g_data )
{
    // write data to global memory
    const unsigned int tid = threadIdx.x;
    int data = g_data[tid];

    // use integer arithmetic to process all four bytes with one thread
    // this serializes the execution, but is the simplest solutions to avoid 
    // bank conflicts for this very low number of threads
    // in general it is more efficient to process each byte by a separate thread,
    // to avoid bank conflicts the access pattern should be 
    // g_data[4 * wtid + wid], where wtid is the thread id within the half warp 
    // and wid is the warp id
    // see also the programming guide for a more in depth discussion.
    g_data[tid] = ((((data <<  0) >> 24) - 10) << 24)
                | ((((data <<  8) >> 24) - 10) << 16)
                | ((((data << 16) >> 24) - 10) <<  8)
                | ((((data << 24) >> 24) - 10) <<  0);
}

///////////////////////////////////////////////////////////////////////////////
//! Demonstration that int2 data can be used in the cpp code
//! @param g_odata  memory to process (in and out)
///////////////////////////////////////////////////////////////////////////////
__global__ void
kernel2( int2* g_data )
{
    // write data to global memory
    const unsigned int tid = threadIdx.x;
    int2 data = g_data[tid];

    // use integer arithmetic to process all four bytes with one thread
    // this serializes the execution, but is the simplest solutions to avoid 
    // bank conflicts for this very low number of threads
    // in general it is more efficient to process each byte by a separate thread,
    // to avoid bank conflicts the access pattern should be 
    // g_data[4 * wtid + wid], where wtid is the thread id within the half warp 
    // and wid is the warp id
    // see also the programming guide for a more in depth discussion.
    g_data[tid].x = data.x - data.y;
}

// Device code
__global__ void VecAdd(int N, float* A, float* B, float* C)
{
	int i = threadIdx.x;
	if (i < N)
	C[i] = A[i] + B[i];
}


__global__ void
corr_cuda( float* C, float* A, float* B, int length, int num /* legnth of vector */)
{
    // Block index
    int x = blockIdx.x * blockDim.x + threadIdx.x;
	if (x >= num){
		return;
	}

	int a=0, b=0;
	float oneoversize = 1.0/length;
	
	

	// a faster version?
	float sum_sq_x, sum_sq_y, sum_coproduct, sweep, i_flt, delta_x, delta_y, mean_x, mean_y;
	float pop_sd_x, pop_sd_y, cov_x_y, p;

	sum_sq_x = 0.0;
	sum_sq_y = 0.0;
	sum_coproduct = 0.0;

	mean_x = A[x*length];
	mean_y = B[0];
	i_flt = (float)2.0;


	// Index of the first vector of A processed by the block
    int aBegin = x*length;
    // Index of the last vector of A processed by the block
    int aEnd   = aBegin + length - 1;
    for (a = aBegin, b = 0;
             a <= aEnd;
             a += 1, b += 1) {
		sweep = ( i_flt - (float)1.0 ) / i_flt;
		delta_x = A[a] - mean_x;
		delta_y = B[b] - mean_y;
		sum_sq_x += delta_x * delta_x * sweep;
		sum_sq_y += delta_y * delta_y * sweep;
		sum_coproduct += delta_x * delta_y * sweep;
		mean_x += delta_x / i_flt;
		mean_y += delta_y / i_flt;
		i_flt += (float)1.0;
	}

	pop_sd_x = sqrt ( sum_sq_x * oneoversize );
	pop_sd_y = sqrt ( sum_sq_y * oneoversize );
	cov_x_y = sum_coproduct * oneoversize;
	
	if ( pop_sd_x * pop_sd_y == 0.0 )
	{
		C[x] = dummy;
	}
	else
	{
		p = cov_x_y / ( pop_sd_x * pop_sd_y );
		C[x] = p;
	}
	
	float min = -1.0;
	float max = 1.0;
	float value = (C[x] - min) / (max - min);
	if(C[x]>max)
		C[x] = 1.0;
	else if(C[x] < min)
		C[x] = 0.0;
	else
		C[x] = value;

/*
    for (int a = aBegin, b = 0;
             a <= aEnd;
             a += 1, b += 1) {
		temp += A[a] -  B[b];
	}
	C[x]=temp;
*/
}

/*
> !--------------------------------------------------------
> ! Number of good (non-missing) data points in the index.
> let idx_ngd = idx_sel[l=@ngd]
> 
> ! Mask the plot variable in time, so that it has exactly the same
> ! missing time points as the anchor index.
> let pvar_mask = IF 1+0*idx_sel THEN pvar_sel
> 
> ! Average of the good data points.
> let idx_ave = idx_sel[l=@sum]/idx_ngd
> let pvar_ave = pvar_mask[l=@sum]/pvar_mask[l=@ngd]
> 
> ! Time series of deviations from the average.
> let idxdev = idx_sel - idx_ave
> let pvardev = pvar_mask - pvar_ave
> 
> ! Time series of cross-products.
> let crossprod = idxdev*pvardev
> 
> ! Time series of squares.
> let idxdev2 = idxdev^2
> let pvardev2 = pvardev^2
> 
> ! Standard deviations.
> let idxstd = (idxdev2[l=@sum]/idx_ngd)^.5
> let pvarstd = (pvardev2[l=@sum]/pvar_mask[l=@ngd])^.5
> 
> ! Covariance, correlation, and regression.
> let cov = crossprod[l=@sum]/crossprod[l=@ngd]
> let corr =  cov/(idxstd*pvarstd)
> let regr =  cov/idxstd^2
> !--------------------------------------------------------
*/

/*
vec4 getPearsonCoefColorLocation ( vec3 texPosition, bool vors, bool bval)
{
	//referenceY = 1 - referenceY;
	texPosition.x = texPosition.x + offsetX;
	if ( texPosition.x<0.0 ) texPosition.x = texPosition.x + 1.0;
	if ( texPosition.x>1.0 ) texPosition.x = texPosition.x - 1.0;

	int i, size = 6;//72
	float oneoversize = 1.0/6.0;//72
	float A[6];//72];
	float B[6];//72]

	float x, y, z;
	float a, zpos, refzpos;
	float zposcp;
	vec3 col;

	if ( grid )
	{
		x = texture1D ( gridTexX, texPosition.x ).r;
		y = texture1D ( gridTexY, texPosition.y ).r;
		z = texPosition.z;
		//z = texture1D(gridTexZ, texPosition.z).r;
	}
	else
	{
		x = texPosition.x;
		y = texPosition.y;
		z = texPosition.z;
	}

	//a = texture3D(volumeTexPearson, vec3(x,y,z)).r;
	//a = 0.05;
	a = 0.01;


	// a faster version?
	float sum_sq_x, sum_sq_y, sum_coproduct, sweep, iflt, delta_x, delta_y, mean_x, mean_y;
	float pop_sd_x, pop_sd_y, cov_x_y, p;

	sum_sq_x = 0.0;
	sum_sq_y = 0.0;
	sum_coproduct = 0.0;

	zpos = z * oneoversize;
	zposcp = zpos;
	refzpos = referenceZ * oneoversize;

	A[0] = lookUpValue ( texture3D ( volumeTex, vec3 ( x,y,zpos ) ).r );
	B[0] = lookUpValue ( texture3D ( volumeTex, vec3 ( referenceX, referenceY, refzpos ) ).r );

	mean_x = A[0];
	mean_y = B[0];
	zpos += oneoversize;
	refzpos += oneoversize;
	iflt = 2.0;


	for ( i=1; i<size; i++ )
	{
		sweep = ( iflt - 1.0 ) / iflt;
		A[i] = lookUpValue ( texture3D ( volumeTex, vec3 ( x,y,zpos ) ).r );
		B[i] = lookUpValue ( texture3D ( volumeTex, vec3 ( referenceX, referenceY, refzpos ) ).r );
		delta_x = A[i] - mean_x;
		delta_y = B[i] - mean_y;
		sum_sq_x += delta_x * delta_x * sweep;
		sum_sq_y += delta_y * delta_y * sweep;
		sum_coproduct += delta_x * delta_y * sweep;
		mean_x += delta_x / iflt;
		mean_y += delta_y / iflt;
		zpos += oneoversize;
		refzpos += oneoversize;
		iflt += 1.0;
	}

	pop_sd_x = sqrt ( sum_sq_x * oneoversize );
	pop_sd_y = sqrt ( sum_sq_y * oneoversize );
	cov_x_y = sum_coproduct * oneoversize;
	if ( pop_sd_x * pop_sd_y == 0.0 )
	{
		return vec4 ( 0.0, 0.0, 0.0, 0.0 );
	}
	else
	{
		p = cov_x_y / ( pop_sd_x * pop_sd_y );
		if(bval){
			return vec4(p, 0.0, 0.0, 0.0);
		}
		col = getCorrelColor ( p, false, vors );
		return vec4 ( col, a );
	}

}
*/




#endif // #ifndef _CPP_INTEGRATION_KERNEL_H_
